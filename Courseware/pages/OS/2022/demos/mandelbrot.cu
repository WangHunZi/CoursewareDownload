
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define MAX_ITER 100
#define DIM 12800
static uint32_t colors[MAX_ITER + 1];
static uint32_t data[DIM * DIM];

__device__ uint32_t mandelbrot(double x, double y) {
  double zr = 0, zi = 0, zrsqr = 0, zisqr = 0;
  int i;

  for (i = 0; i < MAX_ITER; i++) {
    zi = zr * zi * 2 + y;
    zr = zrsqr - zisqr + x;
    zrsqr = zr * zr;
    zisqr = zi * zi;
    if (zrsqr + zisqr > 4.0) {
      break; // SIMT threads diverges here!
    }
  }
  
  return i;
}

__global__ void mandelbrot_kernel(uint32_t *data, double xmin, double ymin, double step, uint32_t *colors) {
  int pix_per_thread = DIM * DIM / (gridDim.x * blockDim.x);
  int tId = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = pix_per_thread * tId;
  for (int i = offset; i < offset + pix_per_thread; i++) {
    int x = i % DIM;
    int y = i / DIM;
    double cr = xmin + x * step;
    double ci = ymin + y * step;
    data[y * DIM + x] = colors[mandelbrot(cr, ci)];
  }
  if (gridDim.x * blockDim.x * pix_per_thread < DIM * DIM
      && tId < (DIM * DIM) - (blockDim.x * gridDim.x)) {
    int i = blockDim.x * gridDim.x * pix_per_thread + tId;
    int x = i % DIM;
    int y = i / DIM;
    double cr = xmin + x * step;
    double ci = ymin + y * step;
    data[y * DIM + x] = colors[mandelbrot(cr, ci)];
  }
}

int main() {
  float freq = 6.3 / MAX_ITER;
  for (int i = 0; i < MAX_ITER; i++) {
    char r = sin(freq * i + 3) * 127 + 128;
    char g = sin(freq * i + 5) * 127 + 128;
    char b = sin(freq * i + 1) * 127 + 128;
    colors[i] = b + 256 * g + 256 * 256 * r;
  }
  colors[MAX_ITER] = 0;

  uint32_t *dev_colors, *dev_data;
  hipMalloc((void**)&dev_colors, sizeof(colors));
  hipMalloc(&dev_data, sizeof(data));
  hipMemcpy(dev_colors, colors, sizeof(colors), hipMemcpyHostToDevice);

  double xcen = -0.5, ycen = 0, scale = 3;
  mandelbrot_kernel<<<512, 512>>>(
    dev_data,
    xcen - (scale / 2),
    ycen - (scale / 2),
    scale / DIM,
    dev_colors
  );

  hipMemcpy(data, dev_data, sizeof(data), hipMemcpyDeviceToHost);
  hipFree(dev_data);
  hipFree(dev_colors);

  FILE *fp = fopen("mandelbrot.ppm", "w");
  fprintf(fp, "P6\n%d %d 255\n", DIM, DIM);
  for (int i = 0; i < DIM * DIM; i++) {
    fputc((data[i] >> 16) & 0xff, fp);
    fputc((data[i] >>  8) & 0xff, fp);
    fputc((data[i] >>  0) & 0xff, fp);
  }
  
  return 0;
}
